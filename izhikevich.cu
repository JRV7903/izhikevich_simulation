// C++ includes

#include <hip/hip_runtime.h>
#include <algorithm>
#include <fstream>

// ------------------------------------------------------------------------
// Helper macro for error-checking CUDA calls
#define CHECK_CUDA_ERRORS(call) {\
    hipError_t error = call;\
    if (error != hipSuccess) {\
        throw std::runtime_error(__FILE__": " + std::to_string(__LINE__) + ": cuda error " + std::to_string(error) + ": " + hipGetErrorString(error));\
    }\
}

namespace
{
unsigned int* glbSpkCntNeurons;
unsigned int* d_glbSpkCntNeurons;
unsigned int* glbSpkNeurons;
unsigned int* d_glbSpkNeurons;

float* VNeurons;
float* d_VNeurons;
float* UNeurons;
float* d_UNeurons;
float* aNeurons;
float* d_aNeurons;
float* bNeurons;
float* d_bNeurons;
float* cNeurons;
float* d_cNeurons;
float* dNeurons;
float* d_dNeurons;

__global__ void preNeuronResetKernel(unsigned int* d_glbSpkCntNeurons) {
    unsigned int id = 32 * blockIdx.x + threadIdx.x;
    if(id == 0) {
        d_glbSpkCntNeurons[0] = 0;
    }
}

__global__ void updateNeuronsKernel(unsigned int *d_glbSpkCntNeurons, unsigned int *d_glbSpkNeurons, float *d_VNeurons, float *d_UNeurons, 
                                    const float *d_aNeurons, const float *d_bNeurons, const float *d_cNeurons, const float *d_dNeurons)
 {
    const unsigned int id = 32 * blockIdx.x + threadIdx.x; 
    __shared__ unsigned int shSpk[32];
    __shared__ unsigned int shPosSpk;
    __shared__ unsigned int shSpkCount;
    if (threadIdx.x == 0); {
        shSpkCount = 0;
    }
    
    __syncthreads();
    // Neurons
    if(id < 32) {
        if(id < 4) {
            float lV = d_VNeurons[id];
            float lU = d_UNeurons[id];
            const float la = d_aNeurons[id];
            const float lb = d_bNeurons[id];
            const float lc = d_cNeurons[id];
            const float ld = d_dNeurons[id];
            
            float Isyn = 0;
            
            // calculate membrane potential
            if (lV >= 30.0f){
                lV=lc;
                lU+=ld;
            } 
            lV+=0.5f*(0.04f*lV*lV+5.0f*lV+140.0f-lU+Isyn+(1.00000000000000000e+01f))*1.0f; //at two times for numerical stability
            lV+=0.5f*(0.04f*lV*lV+5.0f*lV+140.0f-lU+Isyn+(1.00000000000000000e+01f))*1.0f;
            lU+=la*(lb*lV-lU)*1.0f;
            
            if(lV > 30.0){   //keep this to not confuse users with unrealistiv voltage values
              lV=30.0;
            }
            
            // test for and register a true spike
            if (lV >= 29.99f) {
                const unsigned int spkIdx = atomicAdd((unsigned int *) &shSpkCount, 1);
                shSpk[spkIdx] = id;
            }
            d_VNeurons[id] = lV;
            d_UNeurons[id] = lU;
        }
        __syncthreads();
        if (threadIdx.x == 0) {
            if (shSpkCount > 0) {
                shPosSpk = atomicAdd((unsigned int*)&d_glbSpkCntNeurons[0], shSpkCount);
            }
        }
        __syncthreads();
        if (threadIdx.x < shSpkCount) {
            const unsigned int n = shSpk[threadIdx.x];
            d_glbSpkNeurons[shPosSpk + threadIdx.x] = n;
        }
    }
    
}
}

int main()
{
    CHECK_CUDA_ERRORS(hipSetDevice(0));
    
    // Allocate memory
    CHECK_CUDA_ERRORS(hipHostAlloc(&glbSpkCntNeurons, 1 * sizeof(unsigned int), hipHostMallocPortable));
    CHECK_CUDA_ERRORS(hipMalloc(&d_glbSpkCntNeurons, 1 * sizeof(unsigned int)));
    CHECK_CUDA_ERRORS(hipHostAlloc(&glbSpkNeurons, 4 * sizeof(unsigned int), hipHostMallocPortable));
    CHECK_CUDA_ERRORS(hipMalloc(&d_glbSpkNeurons, 4 * sizeof(unsigned int)));
    CHECK_CUDA_ERRORS(hipHostAlloc(&VNeurons, 4 * sizeof(float), hipHostMallocPortable));
    CHECK_CUDA_ERRORS(hipMalloc(&d_VNeurons, 4 * sizeof(float)));
    CHECK_CUDA_ERRORS(hipHostAlloc(&UNeurons, 4 * sizeof(float), hipHostMallocPortable));
    CHECK_CUDA_ERRORS(hipMalloc(&d_UNeurons, 4 * sizeof(float)));
    CHECK_CUDA_ERRORS(hipHostAlloc(&aNeurons, 4 * sizeof(float), hipHostMallocPortable));
    CHECK_CUDA_ERRORS(hipMalloc(&d_aNeurons, 4 * sizeof(float)));
    CHECK_CUDA_ERRORS(hipHostAlloc(&bNeurons, 4 * sizeof(float), hipHostMallocPortable));
    CHECK_CUDA_ERRORS(hipMalloc(&d_bNeurons, 4 * sizeof(float)));
    CHECK_CUDA_ERRORS(hipHostAlloc(&cNeurons, 4 * sizeof(float), hipHostMallocPortable));
    CHECK_CUDA_ERRORS(hipMalloc(&d_cNeurons, 4 * sizeof(float)));
    CHECK_CUDA_ERRORS(hipHostAlloc(&dNeurons, 4 * sizeof(float), hipHostMallocPortable));
    CHECK_CUDA_ERRORS(hipMalloc(&d_dNeurons, 4 * sizeof(float)));
    
    // Initialise
    glbSpkCntNeurons[0] = 0;
    std::fill_n(VNeurons, 4, -6.50000000000000000e+01f);
    std::fill_n(UNeurons, 4, -2.00000000000000000e+01f);
    
    // Regular
    aNeurons[0] = 0.02; bNeurons[0] = 0.2;  cNeurons[0] = -65.0;    dNeurons[0] = 8.0;

    // Fast
    aNeurons[1] = 0.1;  bNeurons[1] = 0.2;  cNeurons[1] = -65.0;    dNeurons[1] = 2.0;

    // Chattering
    aNeurons[2] = 0.02; bNeurons[2] = 0.2;  cNeurons[2] = -50.0;    dNeurons[2] = 2.0;

    // Bursting
    aNeurons[3] = 0.02; bNeurons[3] = 0.2;  cNeurons[3] = -55.0;    dNeurons[3] = 4.0;
    
    // Copy to device
    CHECK_CUDA_ERRORS(hipMemcpy(d_glbSpkCntNeurons, glbSpkCntNeurons, 1 * sizeof(unsigned int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERRORS(hipMemcpy(d_VNeurons, VNeurons, 4 * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERRORS(hipMemcpy(d_UNeurons, UNeurons, 4 * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERRORS(hipMemcpy(d_aNeurons, aNeurons, 4 * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERRORS(hipMemcpy(d_bNeurons, bNeurons, 4 * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERRORS(hipMemcpy(d_cNeurons, cNeurons, 4 * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERRORS(hipMemcpy(d_dNeurons, dNeurons, 4 * sizeof(float), hipMemcpyHostToDevice));
    
    std::ofstream spikes("spikes.csv");
    std::ofstream voltages("voltages.csv");
    
    for(unsigned int i = 0; i < 200; i++) {
        // Launch kernels
        const dim3 threads(32, 1);
        const dim3 grid(1, 1);
        preNeuronResetKernel<<<grid, threads>>>(d_glbSpkCntNeurons);
        updateNeuronsKernel<<<grid, threads>>>(d_glbSpkCntNeurons, d_glbSpkNeurons, d_VNeurons, d_UNeurons, 
                                            d_aNeurons, d_bNeurons, d_cNeurons, d_dNeurons);
        
        // Copy voltages back from device and write to file
        CHECK_CUDA_ERRORS(hipMemcpy(VNeurons, d_VNeurons, 4 * sizeof(float), hipMemcpyDeviceToHost));
        voltages << i << "," << VNeurons[0] << "," << VNeurons[1] << "," << VNeurons[2] << "," << VNeurons[3] << std::endl;
        
        // Copy spikes back from device
        CHECK_CUDA_ERRORS(hipMemcpy(glbSpkCntNeurons, d_glbSpkCntNeurons, 1 * sizeof(unsigned int), hipMemcpyDeviceToHost));
        CHECK_CUDA_ERRORS(hipMemcpy(glbSpkNeurons, d_glbSpkNeurons, 4 * sizeof(unsigned int), hipMemcpyDeviceToHost));
        
        for(unsigned int s = 0; s < glbSpkCntNeurons[0]; s++) {
            spikes << i << "," << glbSpkNeurons[s] << std::endl;
        }

    }
    return EXIT_SUCCESS;
}